#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <ostream>
#include <vector>
#include <random>


#define CELL_LENGTH 30

__device__
int AliveCount(const char* state, int x, int y)
{
    int aliveCount = 0;
    for (int ix = -1; ix <= 1; ++ix)
    {
        for (int iy = -1; iy <= 1; ++iy)
        {
            if (ix == 0 && iy == 0)
            {
                continue;
            }
            if (ix + x < 0 || (ix + x >= CELL_LENGTH))
            {
                continue;
            }
            if (iy + y < 0 || (iy + y >= CELL_LENGTH))
            {
                continue;
            }
            if (state[(x + ix) + (y + iy) * CELL_LENGTH] == 1)
            {
                aliveCount++;
            }
        }
    }
    return aliveCount;
}

__global__
void Compute(const char* state, char* output)
{
    int x = threadIdx.x % CELL_LENGTH;
    int y = threadIdx.x / CELL_LENGTH;
    
    int nbAlive = AliveCount(state, x, y);
    bool isAlive = state[x + y * CELL_LENGTH] == 1;
    if (isAlive)
    {
        if (isAlive && (nbAlive == 3 || nbAlive == 2))
        {
            output[x + y * CELL_LENGTH] = 1;
        }
        else
        {
            output[x + y * CELL_LENGTH] = 0;
        }
    }
    else
    {
        if (!isAlive && nbAlive == 3)
        {
            output[x + y * CELL_LENGTH] = 1;
        }
        else
        {
            output[x + y * CELL_LENGTH] = 0;
        }
    }
}

std::ostream& operator<<(std::ostream& os, const std::vector<char>& state)
{
    for (int x = 0; x < CELL_LENGTH; ++x)
    {
        for (int y = 0; y < CELL_LENGTH; ++y)
        {
            if (state[x + y * CELL_LENGTH] == 1)
            {
                os << " @ ";
            }
            else
            {
                os << "   ";
            }
        }
        os << std::endl;
    }
    return os;
}


int main()
{
    //srand(177013);
    //std::vector<char> State = {};
    //for (int i = 0; i < CELL_LENGTH * CELL_LENGTH; ++i)
    //{
    //    char randValue = (rand() % 2);
    //    State.push_back(randValue);
    //}
    //for (int i = 0; i < 1000; ++i)
    //{
    //    std::cout << "State: " << i << "\n";
    //    std::cout << State;
    //    std::vector<char> newState = Compute(State);
    //    State = newState;
    //    _sleep(10);
    //    std::cout << "\033[H";
    //    //system("cls");
    //	srand(177013);



    //cudification du code


    char* d_A;
    hipMalloc(&d_A, CELL_LENGTH * CELL_LENGTH);

    char* d_outputPtr;
    hipMalloc(&d_outputPtr, CELL_LENGTH * CELL_LENGTH);

    char* h_outputPtr = static_cast<char*>(calloc(CELL_LENGTH * CELL_LENGTH, 1));

    srand(177013);
    std::vector<char> State = {};
    for (int i = 0; i < CELL_LENGTH * CELL_LENGTH; ++i)
    {
        char randValue = (rand() % 2);
        State.push_back(randValue);
    }
    for (int i = 0; i < 1000; ++i)
    {
        std::cout << State;
    	std::cout << "State: " << i;
        
        hipMemcpy(d_A, State.data(), CELL_LENGTH * CELL_LENGTH, hipMemcpyHostToDevice);
        Compute << <1, CELL_LENGTH* CELL_LENGTH >> > (d_A, d_outputPtr);
        hipMemcpy(h_outputPtr, d_outputPtr, State.size(), hipMemcpyDeviceToHost);
        for (int i = 0; i < CELL_LENGTH * CELL_LENGTH; ++i)
        {
            State[i] = h_outputPtr[i];
        }
        _sleep(10);
        std::cout << "\033[H";
        //system("cls");
    }
    hipFree(d_A);
    hipFree(d_outputPtr);
    free(h_outputPtr);
    return 0;
}